// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/phi/kernels/lu_solve_kernel.h"
#include "paddle/phi/backends/dynload/cusolver.h"
#include "paddle/phi/backends/gpu/gpu_context.h"
#include "paddle/phi/backends/gpu/gpu_helper.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/kernel_registry.h"

namespace phi {

template <typename T, typename Context>
void LuSolveKernel(const Context& dev_ctx,
                   const DenseTensor& x,
                   const DenseTensor& lu,
                   const DenseTensor& pivots,
                   const std::string& trans,
                   DenseTensor* out) {
  // Validate input dimensions
  auto x_dims = x.dims();
  auto lu_dims = lu.dims();
  auto pivots_dims = pivots.dims();

  PADDLE_ENFORCE_EQ(lu_dims[lu_dims.size() - 1],
                    lu_dims[lu_dims.size() - 2],
                    phi::errors::InvalidArgument("LU matrix must be square"));

  int n = static_cast<int>(lu_dims[lu_dims.size() - 1]);
  int nrhs = static_cast<int>(x_dims[x_dims.size() - 2]);

  PADDLE_ENFORCE_EQ(
      x_dims[x_dims.size() - 1],
      n,
      phi::errors::InvalidArgument(
          "Dimensions of input matrix and LU matrix do not match"));

  PADDLE_ENFORCE_EQ(pivots_dims[pivots_dims.size() - 1],
                    n,
                    phi::errors::InvalidArgument(
                        "Length of pivots array must equal matrix dimension"));

  dev_ctx.template Alloc<T>(out);

  hipblasOperation_t trans_op;
  if (trans == "N") {
    trans_op = HIPBLAS_OP_N;
  } else if (trans == "T") {
    trans_op = HIPBLAS_OP_T;
  } else if (trans == "C") {
    trans_op = HIPBLAS_OP_C;
  } else {
    PADDLE_THROW(phi::errors::InvalidArgument(
        "trans must be one of ['N', 'T', 'C'], but got %s", trans));
  }

  int lda = n;

  DenseTensor info_tensor;
  info_tensor.Resize({1});
  dev_ctx.template Alloc<int>(&info_tensor);
  int* d_info = info_tensor.data<int>();

  auto handle = dev_ctx.cusolver_dn_handle();

  // Copy x to out since cusolverDn*getrs overwrites the input
  phi::Copy(dev_ctx, x, dev_ctx.GetPlace(), false, out);

  if (std::is_same<T, float>::value) {
    auto* lu_ptr = reinterpret_cast<const float*>(lu.data<T>());
    auto* out_ptr = reinterpret_cast<float*>(out->data<T>());

    PADDLE_ENFORCE_GPU_SUCCESS(dynload::hipsolverDnSgetrs(handle,
                                                         trans_op,
                                                         n,
                                                         nrhs,
                                                         lu_ptr,
                                                         lda,
                                                         pivots.data<int>(),
                                                         out_ptr,
                                                         lda,
                                                         d_info));
  } else if (std::is_same<T, double>::value) {
    auto* lu_ptr = reinterpret_cast<const double*>(lu.data<T>());
    auto* out_ptr = reinterpret_cast<double*>(out->data<T>());

    PADDLE_ENFORCE_GPU_SUCCESS(dynload::hipsolverDnDgetrs(handle,
                                                         trans_op,
                                                         n,
                                                         nrhs,
                                                         lu_ptr,
                                                         lda,
                                                         pivots.data<int>(),
                                                         out_ptr,
                                                         lda,
                                                         d_info));
  }

  // Synchronize to ensure the solve is complete
  dev_ctx.Wait();
}

}  // namespace phi

PD_REGISTER_KERNEL(
    lu_solve, GPU, ALL_LAYOUT, phi::LuSolveKernel, float, double) {}
